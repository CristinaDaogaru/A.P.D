#include "hip/hip_runtime.h"

#include <stdio.h>


#define NUMAR_NODURI 500 

#define NUMAR_MUCHII 500 

#define COST_MAXIM 1000000

typedef struct
{
	int nod1;
	int nod2;
} Muchie;

typedef struct
{
	int nodId;
	bool vizitat;
} Nod;

//Gaseste costul drumului de la nodul start la nodul stop
__device__ __host__ int CautareMuchie(Nod start, Nod stop, Muchie *muchii, int *costuri)
{
	for (int i = 0; i < NUMAR_MUCHII; i++)
		if (muchii[i].nod1 == start.nodId && muchii[i].nod2 == stop.nodId)
			return costuri[i];
	
	return COST_MAXIM;
}

//Cauta nodurile adiacente si calculeaza costul
__global__ void Cauta_Nod(Nod *noduri, Muchie *muchii, int *costuri, int *costTemporal, int *costFinal)
{
	int nod = threadIdx.x;
	if (noduri[nod].vizitat == false)
	{
		noduri[nod].vizitat = true;
		for (int n = 0; n < NUMAR_NODURI; n++)
		{
			//Cauta costul muchiei
			int cost = CautareMuchie(noduri[nod], noduri[n], muchii, costuri);

			//ia costul minim
			if (costFinal[n] > costTemporal[nod] + cost && cost < COST_MAXIM)
				costFinal[n] = costTemporal[nod] + cost;
		}
	}
}

//Retine distanta cea mai mica
__global__ void UpdateCostDrumuri(Nod *noduri, int *costuriTemporale, int *costuriFinale)
{
	int nod = threadIdx.x;
	if (costuriTemporale[nod] > costuriFinale[nod])
	{
		costuriTemporale[nod] = costuriFinale[nod];
		noduri[nod].vizitat = false;
	}
	costuriFinale[nod] = costuriTemporale[nod];
}

void Afisare(int *costuri)
{
	FILE *output = fopen("out.txt", "w");
	for (int i = 0; i < NUMAR_NODURI; i++)
		fprintf(output, "Costul minima la nodul: %d este %d\n", i, costuri[i]);
}

int main()
{
	Nod *noduri;
	Muchie *muchii;
	int *costuri;
	int *costuriMinime, *costuriMinimeTemporale;

	Nod *d_noduri;
	Muchie *d_muchii;
	int *d_costuri;
	int *d_costuriMinime;
	int *d_costuriMinimeTemporale;

	//Creaza timerele

	float runningTime;
	hipEvent_t timeStart, timeEnd;
	hipEventCreate(&timeStart);
	hipEventCreate(&timeEnd);

	//Alocare variabile host
	noduri = (Nod *) malloc(sizeof(Nod) * NUMAR_NODURI);
	muchii = (Muchie *) malloc(sizeof(Muchie) * NUMAR_MUCHII);
	costuri = (int *) malloc(NUMAR_MUCHII * sizeof(int));
	costuriMinime = (int *)malloc(NUMAR_NODURI * sizeof(int));
	costuriMinimeTemporale = (int *)malloc(NUMAR_NODURI * sizeof(int));

	//Alocare variabile device
	hipMalloc((void**)&d_noduri, sizeof(Nod) * NUMAR_NODURI);
	hipMalloc((void**)&d_muchii, sizeof(Muchie) * NUMAR_MUCHII);
	hipMalloc((void**)&d_costuri, NUMAR_MUCHII * sizeof(int));
	hipMalloc((void**)&d_costuriMinime, NUMAR_NODURI * sizeof(int));
	hipMalloc((void**)&d_costuriMinimeTemporale, NUMAR_NODURI * sizeof(int));

	//Initializare vector noduri
	for (int i = 0; i < NUMAR_NODURI; i++)
	{
		Nod a;
		a.nodId = i;
		a.vizitat = false;
		noduri[i] = a;
	}

	//Citire input
	FILE *file = fopen("inp.txt", "r");
	for ( int i = 0; i < NUMAR_MUCHII; i++)
	{
		int nod1, nod2, cost;
		fscanf(file, "%d%d%d", &nod1, &nod2, &cost);
		Muchie e;
		e.nod1 = --nod1;
		e.nod2 = --nod2;
		muchii[i] = e;
		costuri[i] = cost;
	}
	
	//Dijkstra Algorithm
	Nod root = { 0, false };	//nod initial
	root.vizitat = true;

	costuriMinime[root.nodId] = 0;
	costuriMinimeTemporale[root.nodId] = 0;

	//Gaseste drumurile initiale din nodul de start
	for (int i = 0; i < NUMAR_NODURI; i++)
	{
		if (noduri[i].nodId != root.nodId)
		{
			costuriMinime[noduri[i].nodId] = CautareMuchie(root, noduri[i], muchii, costuri);
			costuriMinimeTemporale[noduri[i].nodId] = costuriMinime[noduri[i].nodId];
		}
		else
			noduri[i].vizitat = true;
	}

	//Copiaza variabilele pe device
	hipMemcpy(d_noduri, noduri, sizeof(Nod) * NUMAR_NODURI, hipMemcpyHostToDevice);
	hipMemcpy(d_muchii, muchii, sizeof(Muchie) * NUMAR_MUCHII, hipMemcpyHostToDevice);
	hipMemcpy(d_costuri, costuri, NUMAR_MUCHII * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_costuriMinime, costuriMinime, NUMAR_NODURI * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_costuriMinimeTemporale, costuriMinimeTemporale, NUMAR_NODURI * sizeof(int), hipMemcpyHostToDevice);

	clock_t t;
	t = clock();

	//Executia paralela
		Cauta_Nod <<< 1, NUMAR_NODURI >>>(d_noduri, d_muchii, d_costuri, d_costuriMinime, d_costuriMinimeTemporale);
		//for (int j = 0; j < V; j++)
			UpdateCostDrumuri <<< 1, NUMAR_NODURI >>>(d_noduri, d_costuriMinime, d_costuriMinimeTemporale);
	

	//calculeaza timpul

	t = clock() - t;
	printf("Timpul de executie este: %f ms\n", ((double)t) / CLOCKS_PER_SEC);

	//Copiaza rezultatul inapoi pe device
	hipMemcpy(costuriMinime, d_costuriMinime, NUMAR_NODURI * sizeof(int), hipMemcpyDeviceToHost);

	Afisare(costuriMinime);

	free(noduri);
	free(muchii);
	free(costuri);
	free(costuriMinime);
	free(costuriMinimeTemporale);

	hipFree(d_noduri);
	hipFree(d_muchii);
	hipFree(d_costuri);
	hipFree(d_costuriMinime);
	hipFree(d_costuriMinimeTemporale);

	hipEventDestroy(timeStart);
	hipEventDestroy(timeEnd);
	getchar();
}